#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

using namespace std;

const int N = 1024;
const int BLOCK_SIZE = 256;

__global__ void compute_u(float* A, float* L, float* U, int i) {
    int col = blockIdx.x * blockDim.x + threadIdx.x + i;
    if (col < N) {
        float sum = 0;
        for (int j = 0; j < i; j++) {
            sum += L[i * N + j] * U[j * N + col];
        }
        U[i * N + col] = A[i * N + col] - sum;
    }
}

__global__ void compute_l(float* A, float* L, float* U, int i) {
    int row = blockIdx.x * blockDim.x + threadIdx.x + i + 1;
    if (row < N) {
        float sum = 0;
        for (int j = 0; j < i; j++) {
            sum += L[row * N + j] * U[j * N + i];
        }
        L[row * N + i] = (A[row * N + i] - sum) / U[i * N + i];
    }
}

void print_matrix(const char* name, float* mat, int size) {
    cout << "\n" << name << " (Top " << size << "x" << size << " block):\n";
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%8.2f ", mat[i * N + j]);
        }
        cout << "\n";
    }
}

int main() {
    float *A, *L, *U;
    float *d_A, *d_L, *d_U;

    A = new float[N * N];
    L = new float[N * N];
    U = new float[N * N];

    srand(time(NULL));
    for (int i = 0; i < N * N; i++) {
        A[i] = (float)(rand() % 1000) / 10.0f;
        L[i] = 0.0f;
        U[i] = 0.0f;
    }

    // Set L as identity matrix
    for (int i = 0; i < N; i++) {
        L[i * N + i] = 1.0f;
    }

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_L, N * N * sizeof(float));
    hipMalloc(&d_U, N * N * sizeof(float));

    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_L, L, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, N * N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < N; i++) {
        int blocks = (N - i + BLOCK_SIZE - 1) / BLOCK_SIZE;
        compute_u<<<blocks, BLOCK_SIZE>>>(d_A, d_L, d_U, i);
        if (i < N - 1)
            compute_l<<<blocks, BLOCK_SIZE>>>(d_A, d_L, d_U, i);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(L, d_L, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, N * N * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "\nCUDA LU decomposition time: " << milliseconds / 1000.0f << " seconds\n";

    int print_size = 8;
    print_matrix("Original Matrix A", A, print_size);
    print_matrix("Lower Matrix L", L, print_size);
    print_matrix("Upper Matrix U", U, print_size);

    delete[] A;
    delete[] L;
    delete[] U;

    hipFree(d_A);
    hipFree(d_L);
    hipFree(d_U);

    return 0;
}


/* 
nvcc lup_cuda.cu -O3 -o cuda_lu && ./cuda_lu 
*/