// Simplified CUDA Vector Addition using <ctime>

#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>

constexpr int VECTOR_SIZE = 1'000'000;
constexpr int THREADS_PER_BLOCK = 256;

__global__ void vectorAdd(const int *A, const int *B, int *C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    size_t bytes = VECTOR_SIZE * sizeof(int);
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    hipHostMalloc(&h_A, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_B, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_C, bytes, hipHostMallocDefault);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    for (int i = 0; i < VECTOR_SIZE; ++i) {
        h_A[i] = 1;
        h_B[i] = 2;
    }

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    int blocks = (VECTOR_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    clock_t start = clock();
    vectorAdd<<<blocks, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, VECTOR_SIZE);
    hipDeviceSynchronize();
    clock_t end = clock();

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    bool valid = true;
    for (int i = 0; i < VECTOR_SIZE; ++i) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            valid = false;
            break;
        }
    }

    double ms = 1000.0 * (end - start) / CLOCKS_PER_SEC;
    std::cout << "CUDA execution time: " << ms << " ms\n";
    std::cout << (valid ? "Result is correct!" : "Result is incorrect.") << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
